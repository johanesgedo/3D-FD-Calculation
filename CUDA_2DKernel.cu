#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#define N 1024

#define NX 128
#define NZ 128

#define DX 12.5
#define DZ 12.5
#define DT 0.1

#define BLOCK_DIMX 32
#define BLOCK_DIMY 32
#define sigma1 0.25
#define sigma2 0.75
#define IM 4.0f
#define JM 4.0f

#define p_TM 8
#define p_NF 8

#define PlaneThreads2D \
    for (int j = threadIdx.y; j < p_TM; j += blockDim.y) \
        for (int i = threadIdx.x; i < p_TM; i += blockDim.x)

void checkCUDAError(const char *message) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA Error: %s: %s.\n", message, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__device__ float Galerkin_time[N];
__device__ float Leapfrog_time[N];
__device__ float CrankNicolson_time[N];
__device__ float ADI_time[N];
__device__ float Sigma_time[N];
__device__ float LaxWendroff_time[N];
__device__ float FractionalStep_time[N];
__device__ float MacCormack_time[N];
__device__ float TVD_time[N];
__device__ float PSOR_time[N];
__device__ float FVS_time[N];


__global__ void Galerkin_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start imt
    clock_t start_time = clock();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockDim.y * blockDim.y + threadIdx.y;

    // Ensure the threads are within the grid size
    if (i < nx && j < nz) {
        int idx = i + j * nx;

        // Thread-local input and output arrays
        __shared__ float r_pn[p_NF];    // thread-local input
        __shared__ float r_pp[p_NF];    // thread-locak output

        // Shared memory arrays for second derivatives
        __shared__ float s_d2px[p_TM][p_TM];
        __shared__ float s_d2pz[p_TM][p_TM];

        // Load pressure field per thread memory
        PlaneThreads2D {
            const int idxl = i * p_NF + j * p_TM;
            #pragma unroll
            for (int n = 0; n < p_NF; n++) {
                r_pn[n] = d_pn[idxl + n];
                r_pp[n] = 0.0f;
            }
        }
        __syncthreads();

        // Calculate second derivatives
        PlaneThreads2D {
            const int idxl = i * p_NF + j * p_TM;
            if (i > 0 && i < p_TM - 1) {
                s_d2px[j][i] = (d_pn[idxl + 1] - 2.0f * d_pn[idxl] + d_pn[idxl - 1]) / (dx*dx);
            }
            if (j > 0 && j < p_TM - 1) {
                s_d2pz[j][i] = (d_pn[idxl + p_TM] = 2.0f * d_pn[idxl] + d_pn[idxl - p_TM]) / (dz*dz);
            }
        }
        __syncthreads();

        // compute the wave equation
        PlaneThreads2D {
            const int idxl = i * p_NF + j * p_TM;
            #pragma unroll
            for(int n = 0; n < p_NF; n++) {
                r_pp[n] = d_v[idx] * d_v[idx] * (s_d2px[j][i] + s_d2pz[j][i]) -
                                        (r_pn[n] - 2.0f * d_pn[idxl + n]) / (dt*dt);
            }
        }
        __syncthreads();

        PlaneThreads2D {
            const int idxl = i * p_NF + j * p_TM;
            #pragma unroll
            for (int n = 0; n < p_NF; n++) {
                d_pp[idxl + n] = r_pp[n];
            }
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("Galerkin Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    Galerkin_time[i] = elapsed_time;
}

/// Leapfrog method to solve 2D acoustic wave equation using Micikevisius' algorithm
__global__ void Leapfrog_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iz = blockIdx.y * blockDim.y + threadIdx.y;
    float xscale, zscale;

    xscale = (dt*dt) / (dx*dx);
    zscale = (dt*dt) / (dz*dz);

    if (ix < nx - 12 && iz < nz - 12) {
        int in_idx = ix + 6;
        int out_idx = 0.0f;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz - 6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = (xscale*dx + zscale*dz) * current;

            value += (2 * pow(dz, 2) / 2) *
                            zscale * (infront1 + behind1) +
                     (2 * pow(dx, 2) / 2) *
                            xscale * (s_data[threadIdx.x + 5] +
                                      s_data[threadIdx.x + 7]);

            value += (2 * pow(dz, 4) / 24) *
                            zscale * (infront2 + behind2) +
                     (2 * pow(dx, 4) / 24) *
                            xscale * (s_data[threadIdx.x + 4] +
                                      s_data[threadIdx.x + 8]);

            value += (2 * pow(dz, 6) / 720) *
                            zscale * (infront3 + behind2) +
                     (2 * pow(dx, 6) / 720) *
                            xscale * (s_data[threadIdx.x + 3] +
                                      s_data[threadIdx.x + 9]);

            value += (2 * pow(dz, 8) / 40320) *
                            zscale * (infront4 + behind4) +
                     (2 * pow(dx, 8) / 40320) *
                            xscale * (s_data[threadIdx.x + 2] +
                                      s_data[threadIdx.x + 10]);

            value += (2 * pow(dz, 10) / 3628800) *
                            zscale * (infront5 + behind5) +
                     (2 * pow(dx, 10) / 3628800) *
                            xscale * (s_data[threadIdx.x + 1] +
                                      s_data[threadIdx.x + 11]);

            value += (2 * pow(dz, 12) / 479001600) *
                            zscale * (infront6 + behind6) +
                     (2 * pow(dx, 12) / 479001600) *
                            xscale * (s_data[threadIdx.x + 0] +
                                      s_data[threadIdx.x + 12]);

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("Leapfrog Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    Leapfrog_time[ix] = elapsed_time;
}

__global__ void CrankNicolson_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale, zscale;

    xscale = 0.5 * (dt*dt) / (dx*dx);
    zscale = 0.5 * (dt*dt) / (dz*dz);

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = ((xscale*dx + zscale*dz) +
                           (xscale*(dx+dt) + zscale*(dz+dt))) * current;

            value += ((2 * pow(dz, 2) / 2) + (2 * pow(dt, 2) / 2)) *
                        (2*zscale) * (infront1 + behind1) +
                     ((2 * pow(dx, 2) / 2) *
                        xscale * (s_data[threadIdx.x + 5] +
                                  s_data[threadIdx.x + 7]) +
                      ((2 * pow(dx, 2) / 2) + (2 * pow(dt, 2) / 2)) *
                        xscale * (s_data[threadIdx.x + 5] +
                                  s_data[threadIdx.x + 7]));

            value += ((2 * pow(dz, 4) / 24) + (2 * pow(dt, 4) / 24)) *
                        (2*zscale) * (infront2 + behind2) +
                     ((2 * pow(dx, 4) / 24) *
                        xscale * (s_data[threadIdx.x + 4] +
                                  s_data[threadIdx.x + 8]) +
                      ((2 * pow(dx, 4) / 24) + (2 * pow(dt, 4) / 24)) *
                        xscale * (s_data[threadIdx.x + 4] +
                                  s_data[threadIdx.x + 8]));

            value += ((2 * pow(dz, 6) / 720) + (2 * pow(dt, 6) / 720)) *
                        (2*zscale) * (infront3 + behind3) +
                     ((2 * pow(dx, 6) / 720) *
                        xscale * (s_data[threadIdx.x + 3] +
                                  s_data[threadIdx.x + 9]) +
                      ((2 * pow(dx, 6) / 720) + (2 * pow(dt, 6) / 720)) *
                        xscale * (s_data[threadIdx.x + 3] +
                                  s_data[threadIdx.x + 9]));

            value += ((2 * pow(dz, 8) / 40320) + (2 * pow(dt, 8) / 40320)) *
                        (2*zscale) * (infront4 + behind4) +
                     ((2 * pow(dx, 8) / 40320) *
                        xscale * (s_data[threadIdx.x + 2] +
                                  s_data[threadIdx.x + 10]) +
                      ((2 * pow(dx, 8) / 40320) + (2 * pow(dt, 8) / 40320)) *
                        xscale * (s_data[threadIdx.x + 2] +
                                  s_data[threadIdx.x + 10]));

            value += ((2 * pow(dz, 10) / 3628800) + (2 * pow(dt, 10) / 3628800)) *
                        (2*zscale) * (infront5 + behind5) +
                     ((2 * pow(dx, 10) / 3628800) *
                        xscale * (s_data[threadIdx.x + 1] +
                                  s_data[threadIdx.x + 11]) +
                      ((2 * pow(dx, 10) / 3628800) + (2 * pow(dt, 10) / 3628800)) *
                        xscale * (s_data[threadIdx.x + 1] +
                                  s_data[threadIdx.x + 11]));

            value += ((2 * pow(dz, 12) / 479001600) + (2 * pow(dt, 12) / 479001600)) *
                        (2*zscale) * (infront6 + behind6) +
                     ((2 * pow(dx, 12) / 479001600) *
                        xscale * (s_data[threadIdx.x + 0] +
                                  s_data[threadIdx.x + 12]) +
                      ((2 * pow(dx, 12) / 479001600) + (2 * pow(dt, 12) / 479001600)) *
                        xscale * (s_data[threadIdx.x + 0] +
                                  s_data[threadIdx.x + 12]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("CrankNicolson Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    CrankNicolson_time[ix] = elapsed_time;
}

__global__ void ADI_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale, zscale;
    float dt2 = 0.5*dt;

    xscale = 0.5 * (dt*dt) / (dx*dx);
    zscale = 0.5 * (dt*dt) / (dz*dz);

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = ((xscale*dx + zscale*dz) + (xscale*(dx+dt2) + zscale*(dz+dt))) * current;

            value += ((2 * pow(dz, 2) / 2) + (2 * pow(dt, 2) / 2)) *
                        (2*zscale) * (infront1 + behind1) +
                     ((2 * pow(dx, 2) / 2) *
                        xscale * (s_data[threadIdx.x + 5] +
                                  s_data[threadIdx.x + 7]) +
                      ((2 * pow(dx, 2) / 2) + (2 * pow(dt2, 2) / 2)) *
                        xscale * (s_data[threadIdx.x + 5] +
                                  s_data[threadIdx.x + 7]));

            value += ((2 * pow(dz, 4) / 24) + (2 * pow(dt, 4) / 24)) *
                        (2*zscale) * (infront2 + behind2) +
                     ((2 * pow(dx, 4) / 24) *
                        xscale * (s_data[threadIdx.x + 4] +
                                  s_data[threadIdx.x + 8]) +
                      ((2 * pow(dx, 4) / 24) + (2 * pow(dt2, 4) / 24)) *
                        xscale * (s_data[threadIdx.x + 4] +
                                  s_data[threadIdx.x + 8]));

            value += ((2 * pow(dz, 6) / 720) + (2 * pow(dt, 6) / 720)) *
                        (2*zscale) * (infront3 + behind3) +
                     ((2 * pow(dx, 6) / 720) *
                        xscale * (s_data[threadIdx.x + 3] +
                                  s_data[threadIdx.x + 9]) +
                      ((2 * pow(dx, 6) / 720) + (2 * pow(dt2, 6) / 720)) *
                        xscale * (s_data[threadIdx.x + 3] +
                                  s_data[threadIdx.x + 9]));

            value += ((2 * pow(dz, 8) / 40320) + (2 * pow(dt, 8) / 40320)) *
                        (2*zscale) * (infront4 + behind4) +
                     ((2 * pow(dx, 8) / 40320) *
                        xscale * (s_data[threadIdx.x + 2] +
                                  s_data[threadIdx.x + 10]) +
                      ((2 * pow(dx, 8) / 40320) + (2 * pow(dt2, 8) / 40320)) *
                        xscale * (s_data[threadIdx.x + 2] +
                                  s_data[threadIdx.x + 10]));

            value += ((2 * pow(dz, 10) / 3628800) + (2 * pow(dt, 10) / 3628800)) *
                        (2*zscale) * (infront5 + behind5) +
                     ((2 * pow(dx, 10) / 3628800) *
                        xscale * (s_data[threadIdx.x + 1] +
                                  s_data[threadIdx.x + 11]) +
                      ((2 * pow(dx, 10) / 3628800) + (2 * pow(dt2, 10) / 3628800)) *
                        xscale * (s_data[threadIdx.x + 1] +
                                  s_data[threadIdx.x + 11]));

            value += ((2 * pow(dz, 12) / 479001600) + (2 * pow(dt, 12) / 479001600)) *
                        (2*zscale) * (infront6 + behind6) +
                     ((2 * pow(dx, 12) / 479001600) *
                        xscale * (s_data[threadIdx.x + 0] +
                                  s_data[threadIdx.x + 12]) +
                      ((2 * pow(dx, 12) / 479001600) + (2 * pow(dt2, 12) / 479001600)) *
                        xscale * (s_data[threadIdx.x + 0] +
                                  s_data[threadIdx.x + 12]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("ADI Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    ADI_time[ix] = elapsed_time;
}

__global__ void Sigma_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale1, xscale2, zscale1, zscale2;

    xscale1 = sigma1 * (dt*dt) / (dx*dx);
    xscale2 = sigma2 * (dt*dt) / (dx*dx);

    zscale1 = sigma1 * (dt*dt) / (dz*dz);
    zscale2 = sigma2 * (dt*dt) / (dz*dz);

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-12; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = ((xscale1*(dx+dt) + xscale2*dx) + (zscale1*(dz+dt) + zscale2*dz)) * current;

            value += ((2 * pow(dz, 2) / 2) + (2 * pow(dz, 2) / 2)) *
                        (zscale1 + zscale2) * (infront1 + behind1) +
                     (((2 * pow(dx, 2) / 2) + (2 * pow(dt, 2) / 2)) *
                        xscale1 * (s_data[threadIdx.x + 5] +
                                   s_data[threadIdx.x + 7]) +
                      (2 * pow(dx, 2) / 2) *
                        xscale2 * (s_data[threadIdx.x + 5] +
                                   s_data[threadIdx.x + 7]));

            value += ((2 * pow(dz, 4) / 24) + (2 * pow(dz, 4) / 24)) *
                        (zscale1 + zscale2) * (infront2 + behind2) +
                     (((2 * pow(dx, 4) / 24) + (2 * pow(dt, 4) / 24)) *
                        xscale1 * (s_data[threadIdx.x + 4] +
                                   s_data[threadIdx.x + 8]) +
                      (2 * pow(dx, 4) / 24) *
                        xscale2 * (s_data[threadIdx.x + 4] +
                                   s_data[threadIdx.x + 8]));

            value += ((2 * pow(dz, 6) / 720) + (2 * pow(dz, 6) / 720)) *
                        (zscale1 + zscale2) * (infront3 + behind3) +
                     (((2 * pow(dx, 6) / 720) + (2 * pow(dt, 6) / 720)) *
                        xscale1 * (s_data[threadIdx.x + 3] +
                                   s_data[threadIdx.x + 9]) +
                      (2 * pow(dx, 6) / 720) *
                        xscale2 * (s_data[threadIdx.x + 3] +
                                   s_data[threadIdx.x + 9]));

            value += ((2 * pow(dz, 8) / 40320) + (2 * pow(dz, 8) / 40320)) *
                        (zscale1 + zscale2) * (infront4 + behind4) +
                     (((2 * pow(dx, 8) / 40320) + (2 * pow(dt, 8) / 40320)) *
                        xscale1 * (s_data[threadIdx.x + 2] +
                                   s_data[threadIdx.x + 10]) +
                      (2 * pow(dx, 8) / 40320) *
                        xscale2 * (s_data[threadIdx.x + 2] +
                                   s_data[threadIdx.x + 10]));

            value += ((2 * pow(dz, 10) / 3628800) + (2 * pow(dz, 10) / 3628800)) *
                        (zscale1 + zscale2) * (infront5 + behind5) +
                     (((2 * pow(dx, 10) / 3628800) + (2 * pow(dt, 10) / 3628800)) *
                        xscale1 * (s_data[threadIdx.x + 1] +
                                   s_data[threadIdx.x + 11]) +
                      (2 * pow(dx, 10) / 3628800) *
                        xscale2 * (s_data[threadIdx.x + 1] +
                                   s_data[threadIdx.x + 11]));

            value += ((2 * pow(dz, 12) / 479001600) + (2 * pow(dz, 12) / 479001600)) *
                        (zscale1 + zscale2) * (infront6 + behind6) +
                     (((2 * pow(dx, 12) / 479001600) + (2 * pow(dt, 12) / 479001600)) *
                        xscale1 * (s_data[threadIdx.x + 0] +
                                   s_data[threadIdx.x + 12]) +
                      (2 * pow(dx, 12) / 479001600) *
                        xscale2 * (s_data[threadIdx.x + 0] +
                                   s_data[threadIdx.x + 12]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("Sigma Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    Sigma_time[ix] = elapsed_time;
}

__global__ void LaxWendroff_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale_courant, xscale_diffusion;
    float zscale_courant, zscale_diffusion;

    xscale_courant = 0.5 * dt / (dx*dx);
    xscale_diffusion = 0.5 * (dt*dt) / (dx*dx);

    zscale_courant = 0.5 * dt / (dz*dz);
    zscale_diffusion = 0.5 * (dt*dt) / (dz*dz);

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = ((xscale_diffusion*dx + zscale_diffusion*dz) +
                           (xscale_courant*dx + zscale_courant*dz)) * current;

            value += ((2 * pow(dz, 2) / 2) + (2 * pow(dz, 1) / 1)) *
                        (zscale_courant+zscale_diffusion) * (infront1 + behind1) +
                     ((2 * pow(dx, 1) / 1) *
                        xscale_courant * (s_data[threadIdx.x + 5] +
                                          s_data[threadIdx.x + 7]) +
                      (2 * pow(dx, 2) / 2) *
                        xscale_diffusion * (s_data[threadIdx.x + 5] +
                                            s_data[threadIdx.x + 7]));

            value += ((2 * pow(dz, 4) / 24) + (2 * pow(dz, 3) / 6)) *
                        (zscale_courant+zscale_diffusion) * (infront2 + behind2) +
                     ((2 * pow(dx, 3) / 6) *
                        xscale_courant * (s_data[threadIdx.x + 4] +
                                          s_data[threadIdx.x + 8]) +
                      (2 * pow(dx, 4) / 24) *
                        xscale_diffusion * (s_data[threadIdx.x + 4] +
                                            s_data[threadIdx.x + 8]));

            value += ((2 * pow(dz, 6) / 720) + (2 * pow(dz, 5) / 120)) *
                        (zscale_courant+zscale_diffusion) * (infront3 + behind3) +
                     ((2 * pow(dx, 5) / 120) *
                        xscale_courant * (s_data[threadIdx.x + 3] +
                                          s_data[threadIdx.x + 9]) +
                      (2 * pow(dx, 6) / 720) *
                        xscale_diffusion * (s_data[threadIdx.x + 3] +
                                            s_data[threadIdx.x + 9]));

            value += ((2 * pow(dz, 8) / 40320) + (2 * pow(dz, 7) / 5040)) *
                        (zscale_courant+zscale_diffusion) * (infront4 + behind4) +
                     ((2 * pow(dx, 7) / 5040) *
                        xscale_courant * (s_data[threadIdx.x + 2] +
                                          s_data[threadIdx.x + 10]) +
                      (2 * pow(dx, 8) / 40320) *
                        xscale_diffusion * (s_data[threadIdx.x + 2] +
                                            s_data[threadIdx.x + 10]));

            value += ((2 * pow(dz, 10) / 3628800) + (2 * pow(dz, 9) / 362880)) *
                        (zscale_courant+zscale_diffusion) * (infront5 + behind5) +
                     ((2 * pow(dx, 9) / 5040) *
                        xscale_courant * (s_data[threadIdx.x + 1] +
                                          s_data[threadIdx.x + 11]) +
                      (2 * pow(dx, 10) / 3628800) *
                        xscale_diffusion * (s_data[threadIdx.x + 1] +
                                            s_data[threadIdx.x + 11]));

            value += ((2 * pow(dz, 12) / 479001600) + (2 * pow(dz, 11) / 39916800)) *
                        (zscale_courant+zscale_diffusion) * (infront6 + behind6) +
                     ((2 * pow(dx, 11) / 39916800) *
                        xscale_courant * (s_data[threadIdx.x + 0] +
                                          s_data[threadIdx.x + 12]) +
                      (2 * pow(dx, 12) / 479001600) *
                        xscale_diffusion * (s_data[threadIdx.x + 0] +
                                            s_data[threadIdx.x + 12]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("LaxWendroff Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    LaxWendroff_time[ix] = elapsed_time;
}

__global__ void FractionalStep_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale, zscale;
    float dt2 = 0.5*dt;

    xscale = 0.5 * (dt*dt*0.5) / (dx*dx);
    zscale = 0.5 * (dt*dt*0.5) / (dz*dz);

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = ((xscale*(dx+dt2) + zscale*(dz+dt)) + (xscale*dx + zscale*dz)) * current;

            value += ((2 * pow(dz, 2) / 2) + (2 * pow(dt, 2) / 2)) *
                        (4*zscale) * (infront1 + behind1) +
                     (((2 * pow(dx, 2) / 2) + (2 * pow(dt2, 2) / 2)) *
                        (2*xscale) * (s_data[threadIdx.x + 5] +
                                      s_data[threadIdx.x + 7]) +
                      (2 * pow(dx, 2) / 2) *
                        (2*xscale) * (s_data[threadIdx.x + 5] +
                                      s_data[threadIdx.x + 7]));

            value += ((2 * pow(dz, 4) / 24) + (2 * pow(dt, 4) / 24)) *
                        (4*zscale) * (infront2 + behind2) +
                     (((2 * pow(dx, 4) / 24) + (2 * pow(dt2, 4) / 24)) *
                        (2*xscale) * (s_data[threadIdx.x + 4] +
                                      s_data[threadIdx.x + 8]) +
                      (2 * pow(dx, 4) / 24) *
                        (2*xscale) * (s_data[threadIdx.x + 4] +
                                      s_data[threadIdx.x + 8]));

            value += ((2 * pow(dz, 6) / 720) + (2 * pow(dt, 6) / 720)) *
                        (4*zscale) * (infront3 + behind3) +
                     (((2 * pow(dx, 6) / 720) + (2 * pow(dt2, 6) / 720)) *
                        (2*xscale) * (s_data[threadIdx.x + 3] +
                                      s_data[threadIdx.x + 9]) +
                      (2 * pow(dx, 6) / 720) *
                        (2*xscale) * (s_data[threadIdx.x + 3] +
                                      s_data[threadIdx.x + 9]));

            value += ((2 * pow(dz, 8) / 40320) + (2 * pow(dt, 8) / 40320)) *
                        (4*zscale) * (infront4 + behind4) +
                     (((2 * pow(dx, 8) / 40320) + (2 * pow(dt2, 8) / 40320)) *
                        (2*xscale) * (s_data[threadIdx.x + 2] +
                                      s_data[threadIdx.x + 10]) +
                      (2 * pow(dx, 8) / 40320) *
                        (2*xscale) * (s_data[threadIdx.x + 2] +
                                      s_data[threadIdx.x + 10]));

            value += ((2 * pow(dz, 10) / 3628800) + (2 * pow(dt, 10) / 3628800)) *
                        (4*zscale) * (infront5 + behind5) +
                     (((2 * pow(dx, 10) / 3628800) + (2 * pow(dt2, 10) / 3628800)) *
                        (2*xscale) * (s_data[threadIdx.x + 1] +
                                      s_data[threadIdx.x + 11]) +
                      (2 * pow(dx, 10) / 3628800) *
                        (2*xscale) * (s_data[threadIdx.x + 1] +
                                      s_data[threadIdx.x + 11]));

            value += ((2 * pow(dz, 12) / 479001600) + (2 * pow(dt, 12) / 479001600)) *
                        (4*zscale) * (infront6 + behind6) +
                     (((2 * pow(dx, 12) / 479001600) + (2 * pow(dt2, 12) / 479001600)) *
                        (2*xscale) * (s_data[threadIdx.x + 0] +
                                      s_data[threadIdx.x + 12]) +
                      (2 * pow(dx, 12) / 479001600) *
                        (2*xscale) * (s_data[threadIdx.x + 0] +
                                      s_data[threadIdx.x + 12]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("FractionalStep Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    FractionalStep_time[ix] = elapsed_time;
}

__global__ void MacCormack_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale_predictor, xscale_corrector;
    float zscale_predictor, zscale_corrector;

    xscale_predictor = 0.5 * dt / (dx*dx);
    xscale_corrector = (dt*dt) / (dx*dx);

    zscale_predictor = 0.5 * dt / (dz*dz);
    zscale_corrector = (dt*dt) / (dz*dz);

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = ((xscale_corrector*(dx+dt) + zscale_corrector*(dz+dt)) +
                           (xscale_predictor*dx + zscale_predictor*dz)) * current;

            value += (((2 * pow(dz, 2) / 2) + (2 * pow(dt, 2) / 2)) *
                        zscale_corrector * (infront1 + behind1) +
                      (2 * pow(dz, 1) / 1) *
                        zscale_predictor * (infront1 + behind1)) +
                     (((2 * pow(dx, 2) / 2) + (2 * pow(dt, 2) / 2)) *
                        xscale_corrector * (s_data[threadIdx.x + 5] +
                                            s_data[threadIdx.x + 7]) +
                      (2 * pow(dx, 1) / 1) *
                        xscale_predictor * (s_data[threadIdx.x + 5] +
                                            s_data[threadIdx.x + 7]));

            value += (((2 * pow(dz, 4) / 24) + (2 * pow(dt, 4) / 24)) *
                        zscale_corrector * (infront2 + behind2) +
                      (2 * pow(dz, 3) / 6) *
                        zscale_predictor * (infront2 + behind2)) +
                     (((2 * pow(dx, 4) / 24) + (2 * pow(dt, 4) / 24)) *
                        xscale_corrector * (s_data[threadIdx.x + 4] +
                                            s_data[threadIdx.x + 8]) +
                      (2 * pow(dx, 3) / 6) *
                        xscale_predictor * (s_data[threadIdx.x + 4] +
                                            s_data[threadIdx.x + 8]));

            value += (((2 * pow(dz, 6) / 720) + (2 * pow(dt, 6) / 720)) *
                        zscale_corrector * (infront3 + behind3) +
                      (2 * pow(dz, 5) / 120) *
                        zscale_predictor * (infront3 + behind3)) +
                     (((2 * pow(dx, 6) / 720) + (2 * pow(dt, 6) / 720)) *
                        xscale_corrector * (s_data[threadIdx.x + 3] +
                                            s_data[threadIdx.x + 9]) +
                      (2 * pow(dx, 5) / 120) *
                        xscale_predictor * (s_data[threadIdx.x + 3] +
                                            s_data[threadIdx.x + 9]));

            value += (((2 * pow(dz, 8) / 40320) + (2 * pow(dt, 8) / 40320)) *
                        zscale_corrector * (infront4 + behind4) +
                      (2 * pow(dz, 7) / 5040) *
                        zscale_predictor * (infront4 + behind4)) +
                     (((2 * pow(dx, 8) / 40320) + (2 * pow(dt, 8) / 40320)) *
                        xscale_corrector * (s_data[threadIdx.x + 2] +
                                            s_data[threadIdx.x + 10]) +
                      (2 * pow(dx, 7) / 5040) *
                        xscale_predictor * (s_data[threadIdx.x + 2] +
                                            s_data[threadIdx.x + 10]));

            value += (((2 * pow(dz, 10) / 3628800) + (2 * pow(dt, 10) / 3628800)) *
                        zscale_corrector * (infront5 + behind5) +
                      (2 * pow(dz, 9) / 362880) *
                        zscale_predictor * (infront5 + behind5)) +
                     (((2 * pow(dx, 10) / 3628800) + (2 * pow(dt, 10) / 3628800)) *
                        xscale_corrector * (s_data[threadIdx.x + 1] +
                                            s_data[threadIdx.x + 11]) +
                      (2 * pow(dx, 9) / 362880) *
                        xscale_predictor * (s_data[threadIdx.x + 1] +
                                            s_data[threadIdx.x + 11]));

            value += (((2 * pow(dz, 12) / 479001600) + (2 * pow(dt, 12) / 479001600)) *
                        zscale_corrector * (infront6 + behind6) +
                      (2 * pow(dz, 11) / 39916800) *
                        zscale_predictor * (infront6 + behind6)) +
                     (((2 * pow(dx, 12) / 479001600) + (2 * pow(dt, 12) / 479001600)) *
                        xscale_corrector * (s_data[threadIdx.x + 0] +
                                            s_data[threadIdx.x + 12]) +
                      (2 * pow(dx, 11) / 39916800) *
                        xscale_predictor * (s_data[threadIdx.x + 0] +
                                            s_data[threadIdx.x + 12]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("MacCormack Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    MacCormack_time[ix] = elapsed_time;
}

__global__ void TVD_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale_first, xscale_second, zscale_first, zscale_second;

    xscale_first = 0.5 * dt / dx;
    xscale_second = (dt*dt) / (dx*dx);

    zscale_first = 0.5 * dt / dz;
    zscale_second = (dt*dt) / (dz*dz);

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        /// Define index thread
        int tx = threadIdx.x + 6;

        /// Symmetric Flux Limiters from Waterson & Deconinck, 2007.
        /// Paper: "Design principles for bounded higher-order convection schemes   a unified approach"
        /// int rx = (tx-1)/(tx+1);
        /// int flx = (1.5*(pow(rx,2)+rx)) / (pow(rx,2)+rx+1);
        int flx1 = (1.5*(pow((tx-1)/(tx+1),2)+((tx-1)/(tx+1)))) / (pow((tx-1)/(tx+1),2)+((tx-1)/(tx+1))+1);
        int flx2 = (1.5*(pow((tx-2)/(tx+2),2)+((tx-2)/(tx+2)))) / (pow((tx-2)/(tx+2),2)+((tx-2)/(tx+2))+1);
        int flx3 = (1.5*(pow((tx-3)/(tx+4),2)+((tx-3)/(tx+3)))) / (pow((tx-3)/(tx+3),2)+((tx-3)/(tx+3))+1);
        int flx4 = (1.5*(pow((tx-4)/(tx+4),2)+((tx-4)/(tx+4)))) / (pow((tx-4)/(tx+4),2)+((tx-4)/(tx+4))+1);
        int flx5 = (1.5*(pow((tx-5)/(tx+5),2)+((tx-5)/(tx+5)))) / (pow((tx-5)/(tx+5),2)+((tx-5)/(tx+5))+1);
        int flx6 = (1.5*(pow((tx-6)/(tx+6),2)+((tx-6)/(tx+6)))) / (pow((tx-6)/(tx+6),2)+((tx-6)/(tx+6))+1);

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx  += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[tx] = current;
            __syncthreads();

            float value = ((xscale_second*dx + zscale_second*dz) +
                           (xscale_first*dx + zscale_first*dz)) * current;

            value += ((2 * pow(dz, 1) / 1) *
                        zscale_first * (infront1 + behind1) +
                      (2 * pow(dz, 2) / 2) *
                        zscale_second * (infront1 + behind1)) +
                     ((2 * pow(dx, 1) / 1) *
                        xscale_first * (s_data[tx - 1 - flx1] +
                                        s_data[tx + 1 + flx1]) +
                      (2 * pow(dx, 2) / 2) *
                        xscale_second * (s_data[tx - 1] +
                                         s_data[tx + 1]));

            value += ((2 * pow(dz, 3) / 6) *
                        zscale_first * (infront2 + behind2) +
                      (2 * pow(dz, 4) / 24) *
                        zscale_second * (infront2 + behind2)) +
                     ((2 * pow(dx, 3) / 6) *
                        xscale_first * (s_data[tx - 2 - flx2] +
                                        s_data[tx + 2 + flx2]) +
                      (2 * pow(dx, 4) / 24) *
                        xscale_second * (s_data[tx - 2] +
                                         s_data[tx + 2]));

            value += ((2 * pow(dz, 5) / 120) *
                        zscale_first * (infront3 + behind3) +
                      (2 * pow(dz, 6) / 720) *
                        zscale_second * (infront3 + behind3)) +
                     ((2 * pow(dx, 5) / 120) *
                        xscale_first * (s_data[tx - 3 - flx3] +
                                        s_data[tx + 3 + flx3]) +
                      (2 * pow(dx, 6) / 720) *
                        xscale_second * (s_data[tx - 3] +
                                         s_data[tx + 3]));

            value += ((2 * pow(dz, 7) / 5040) *
                        zscale_first * (infront4 + behind4) +
                      (2 * pow(dz, 8) / 40320) *
                        zscale_second * (infront4 + behind4)) +
                     ((2 * pow(dx, 7) / 5040) *
                        xscale_first * (s_data[tx - 4 - flx4] +
                                        s_data[tx + 4 + flx4]) +
                      (2 * pow(dx, 8) / 40320) *
                        xscale_second * (s_data[tx - 4] +
                                         s_data[tx + 4]));

            value += ((2 * pow(dz, 9) / 362880) *
                        zscale_first * (infront5 + behind5) +
                      (2 * pow(dz, 10) / 3628800) *
                        zscale_second * (infront5 + behind5)) +
                     ((2 * pow(dx, 9) / 362880) *
                        xscale_first * (s_data[tx - 5 - flx5] +
                                        s_data[tx + 5 + flx5]) +
                      (2 * pow(dx, 10) / 3628800) *
                        xscale_second * (s_data[tx - 5] +
                                         s_data[tx + 5]));

            value += ((2 * pow(dz, 11) / 39916800) *
                        zscale_first * (infront6 + behind6) +
                      (2 * pow(dz, 12) / 479001600) *
                        zscale_second * (infront6 + behind6)) +
                     ((2 * pow(dx, 11) / 39916800) *
                        xscale_first * (s_data[tx - 6 - flx6] +
                                        s_data[tx + 6 + flx6]) +
                      (2 * pow(dx, 12) / 479001600) *
                        xscale_second * (s_data[tx - 6] +
                                         s_data[tx + 6]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("TVD Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    TVD_time[ix] = elapsed_time;
}

__global__ void PSOR_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale, zscale;
    float first_scale, second_scale, sor, a, wopt;

    xscale = (dt*dt) / (dx*dx);
    zscale = (dt*dt) / (dz*dz);

    /// Based on Hoffmann(2000) approximation
    sor = xscale / zscale;
    a = powf((cos(M_PI/(IM-1)) + (sor*sor)*cos(M_PI/(JM-1))) / (1+sor*sor), 2);
    wopt = (2-2*sqrt(1-a))/a;

    first_scale = wopt / (2*(1+sor*sor));
    second_scale = wopt*(sor*sor) / (2*(1+sor*sor));

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        float coefficient_1 = dx + dz;
        float coefficient_2 = 2*pow(dx,2)/2 + 2*pow(dz,2)/2;
        float coefficient_3 = 2*pow(dx,4)/24 + 2*pow(dz,4)/24;
        float coefficient_4 = 2*pow(dx,6)/720 + 2*pow(dz,6)/720;
        float coefficient_5 = 2*pow(dx,8)/40320 + 2*pow(dz,8)/40320;
        float coefficient_6 = 2*pow(dx,10)/3628800 + 2*pow(dz,10)/3628800;
        float coefficient_7 = 2*pow(dx,12)/479001600 + 2*pow(dz,12)/479001600;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = ((first_scale+second_scale) * wopt) * (coefficient_1) * current;

            value += coefficient_2 * (first_scale * (infront1 + behind1) +
                                      second_scale * (s_data[threadIdx.x + 5] +
                                                      s_data[threadIdx.x + 7]));

            value += coefficient_3 * (first_scale * (infront2 + behind2) +
                                      second_scale * (s_data[threadIdx.x + 4] +
                                                      s_data[threadIdx.x + 8]));

            value += coefficient_4 * (first_scale * (infront3 + behind3) +
                                      second_scale * (s_data[threadIdx.x + 3] +
                                                      s_data[threadIdx.x + 9]));

            value += coefficient_5 * (first_scale * (infront4 + behind4) +
                                      second_scale * (s_data[threadIdx.x + 2] +
                                                      s_data[threadIdx.x + 10]));

            value += coefficient_6 * (first_scale * (infront5 + behind5) +
                                      second_scale * (s_data[threadIdx.x + 1] +
                                                      s_data[threadIdx.x + 11]));

            value += coefficient_7 * (first_scale * (infront6 + behind6) +
                                      second_scale * (s_data[threadIdx.x + 0] +
                                                      s_data[threadIdx.x + 12]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("PSOR Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    PSOR_time[ix] = elapsed_time;
}

__global__ void FVS_2D_Solver(
    int nx, float dx,
    int nz, float dz, float dt,
    float* __restrict__ d_v,
    float* __restrict__ d_pn,
    float* __restrict__ d_pp
)
{
    // Get the start time
    clock_t start_time = clock();

    __shared__ float s_data[BLOCK_DIMX + 12];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iz = blockDim.y * blockIdx.y + threadIdx.y;
    float xscale_one, xscale_two, zscale_one, zscale_two;

    xscale_one = 0.5 * dt / dx;
    xscale_two = 0.5 * (dt*dt) / (dx*dx);

    zscale_one = 0.5 * dt / dz;
    zscale_two = 0.5 * (dt*dt) / (dz*dz);

    if (ix < nx-12 && iz < nz-12) {
        int in_idx = ix + 6;
        int out_idx = 0;
        int stride = nx;

        float infront1, infront2, infront3, infront4, infront5, infront6;
        float behind1, behind2, behind3, behind4, behind5, behind6;
        float current;

        behind5 = d_pn[in_idx];
        in_idx += stride;
        behind4 = d_pn[in_idx];
        in_idx += stride;
        behind3 = d_pn[in_idx];
        in_idx += stride;
        behind2 = d_pn[in_idx];
        in_idx += stride;
        behind1 = d_pn[in_idx];
        in_idx += stride;

        current = d_pn[in_idx];
        out_idx = in_idx;
        in_idx += stride;

        infront1 = d_pn[in_idx];
        in_idx += stride;
        infront2 = d_pn[in_idx];
        in_idx += stride;
        infront3 = d_pn[in_idx];
        in_idx += stride;
        infront4 = d_pn[in_idx];
        in_idx += stride;
        infront5 = d_pn[in_idx];
        in_idx += stride;
        infront6 = d_pn[in_idx];
        in_idx += stride;

#pragma unroll
        for (iz = 6; iz < nz-6; iz++) {
            behind6 = behind5;
            behind5 = behind4;
            behind4 = behind3;
            behind3 = behind2;
            behind2 = behind1;
            behind1 = current;
            current = infront1;
            infront1 = infront2;
            infront2 = infront3;
            infront3 = infront4;
            infront4 = infront5;
            infront5 = infront6;
            infront6 = d_pn[in_idx];

            in_idx += stride;
            out_idx += stride;

            __syncthreads();

            if (threadIdx.x < 6) {
                s_data[threadIdx.x] = d_pn[out_idx - 6];
                s_data[threadIdx.x + BLOCK_DIMX + 6] = d_pn[out_idx + BLOCK_DIMX];
            }

            s_data[threadIdx.x + 6] = current;
            __syncthreads();

            float value = ((xscale_one*dx + zscale_one*dz) +
                           (xscale_two*dx + zscale_two*dz)) * current;

            value += ((2 * pow(dz, 1) / 1) *
                        zscale_one * (infront1 - behind1) +
                      (2 * pow(dz, 2) / 2) *
                        zscale_two * (infront1 + behind1)) +
                     ((2 * pow(dx, 1) / 1) *
                        xscale_one * (s_data[threadIdx.x + 5] -
                                      s_data[threadIdx.x + 7]) +
                      (2 * pow(dx, 2) / 2) *
                        xscale_two * (s_data[threadIdx.x + 5] +
                                      s_data[threadIdx.x + 7]));

            value += ((2 * pow(dz, 3) / 6) *
                        zscale_one * (infront2 - behind2) +
                      (2 * pow(dz, 4) / 24) *
                        zscale_two * (infront2 + behind2)) +
                     ((2 * pow(dx, 3) / 6) *
                        xscale_one * (s_data[threadIdx.x + 4] -
                                      s_data[threadIdx.x + 8]) +
                      (2 * pow(dx, 4) / 24) *
                        xscale_two * (s_data[threadIdx.x + 4] +
                                      s_data[threadIdx.x + 8]));

            value += ((2 * pow(dz, 5) / 120) *
                        zscale_one * (infront3 - behind3) +
                      (2 * pow(dz, 6) / 720) *
                        zscale_two * (infront3 + behind3)) +
                     ((2 * pow(dx, 5) / 120) *
                        xscale_one * (s_data[threadIdx.x + 3] -
                                      s_data[threadIdx.x + 9]) +
                      (2 * pow(dx, 6) / 720) *
                        xscale_two * (s_data[threadIdx.x + 3] +
                                      s_data[threadIdx.x + 9]));

            value += ((2 * pow(dz, 7) / 5040) *
                        zscale_one * (infront4 - behind4) +
                      (2 * pow(dz, 8) / 40320) *
                        zscale_two * (infront4 + behind4)) +
                     ((2 * pow(dx, 7) / 5040) *
                        xscale_one * (s_data[threadIdx.x + 2] -
                                      s_data[threadIdx.x + 10]) +
                      (2 * pow(dx, 8) / 40320) *
                        xscale_two * (s_data[threadIdx.x + 2] +
                                      s_data[threadIdx.x + 10]));

            value += ((2 * pow(dz, 9) / 362880) *
                        zscale_one * (infront5 - behind5) +
                      (2 * pow(dz, 10) / 3628800) *
                        zscale_two * (infront5 + behind5)) +
                     ((2 * pow(dx, 9) / 362880) *
                        xscale_one * (s_data[threadIdx.x + 1] -
                                      s_data[threadIdx.x + 11]) +
                      (2 * pow(dx, 10) / 3628800) *
                        xscale_two * (s_data[threadIdx.x + 1] +
                                      s_data[threadIdx.x + 11]));

            value += ((2 * pow(dz, 11) / 39916800) *
                        zscale_one * (infront6 - behind6) +
                      (2 * pow(dz, 12) / 479001600) *
                        zscale_two * (infront6 + behind6)) +
                     ((2 * pow(dx, 11) / 39916800) *
                        xscale_one * (s_data[threadIdx.x + 0] -
                                      s_data[threadIdx.x + 12]) +
                      (2 * pow(dx, 12) / 479001600) *
                        xscale_two * (s_data[threadIdx.x + 0] +
                                      s_data[threadIdx.x + 12]));

            d_pp[out_idx] = 2.0f * current - d_pp[out_idx] + d_v[out_idx] * value;
        }
    }

    // Get the end time
    clock_t end_time = clock();

    // Calculate the elapsed time in milliseconds
    float elapsed_time = 1000.0 * (float)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print the elapsed time
    // printf("FVS Execution time: %f ms\n", elapsed_time);

    // Save the elapsed time to the global array
    FVS_time[ix] = elapsed_time;
}


int main (int argc, char **argv) {
    // Set problem size
    int nx = NX;
    int nz = NZ;
    int size = nx * nz;

    // Set simulation parameters
    float dx = DX;
    float dz = DZ;
    float dt = DT;

    // Allocate memory on the host
    float *h_pn_Galerkin = (float*)malloc(size * sizeof(float));
    float *h_pn_Leapfrog = (float*)malloc(size * sizeof(float));
    float *h_pn_CrankNicolson = (float*)malloc(size * sizeof(float));
    float *h_pn_ADI = (float*)malloc(size * sizeof(float));
    float *h_pn_Sigma = (float*)malloc(size * sizeof(float));
    float *h_pn_LaxWendroff = (float*)malloc(size * sizeof(float));
    float *h_pn_FractionalStep = (float*)malloc(size * sizeof(float));
    float *h_pn_MacCormack = (float*)malloc(size * sizeof(float));
    float *h_pn_TVD = (float*)malloc(size * sizeof(float));
    float *h_pn_PSOR = (float*)malloc(size * sizeof(float));
    float *h_pn_FVS = (float*)malloc(size * sizeof(float));
    float *h_v = (float*)malloc(size * sizeof(float));
    float *h_pp_Galerkin = (float*)malloc(size * sizeof(float));
    float *h_pp_Leapfrog = (float*)malloc(size * sizeof(float));
    float *h_pp_CrankNicolson = (float*)malloc(size * sizeof(float));
    float *h_pp_ADI = (float*)malloc(size * sizeof(float));
    float *h_pp_Sigma = (float*)malloc(size * sizeof(float));
    float *h_pp_LaxWendroff = (float*)malloc(size * sizeof(float));
    float *h_pp_FractionalStep = (float*)malloc(size * sizeof(float));
    float *h_pp_MacCormack = (float*)malloc(size * sizeof(float));
    float *h_pp_TVD = (float*)malloc(size * sizeof(float));
    float *h_pp_PSOR = (float*)malloc(size * sizeof(float));
    float *h_pp_FVS = (float*)malloc(size * sizeof(float));

    // Initialize input data with random values
#pragma unroll
    for (int i=0; i < size; i++) {
	    h_pn_Galerkin[i] = h_pn_Leapfrog[i] = h_pn_CrankNicolson[i] = h_pn_ADI[i] =
	    h_pn_Sigma[i] = h_pn_LaxWendroff[i] = h_pn_FractionalStep[i] = h_pn_TVD[i] =
	    h_pn_MacCormack[i] = h_pn_PSOR[i] = h_pn_FVS[i] = 1 + rand() % 1000;
    }

    // Allocate memory on the device
    float *d_pn_Galerkin, *d_pn_Leapfrog, *d_pn_CrankNicolson, *d_pn_ADI;
    float *d_pn_Sigma, *d_pn_LaxWendroff, *d_pn_FractionalStep, *d_pn_MacCormack;
    float *d_pn_TVD, *d_pn_PSOR, *d_pn_FVS;
    float *d_v;
    float *d_pp_Galerkin, *d_pp_Leapfrog, *d_pp_CrankNicolson, *d_pp_ADI;
    float *d_pp_Sigma, *d_pp_LaxWendroff, *d_pp_FractionalStep, *d_pp_MacCormack;
    float *d_pp_TVD, *d_pp_PSOR, *d_pp_FVS;
    hipMalloc((void**)&d_pn_Galerkin, size * sizeof(float));
    hipMalloc((void**)&d_pn_Leapfrog, size * sizeof(float));
    hipMalloc((void**)&d_pn_CrankNicolson, size * sizeof(float));
    hipMalloc((void**)&d_pn_ADI, size * sizeof(float));
    hipMalloc((void**)&d_pn_Sigma, size * sizeof(float));
    hipMalloc((void**)&d_pn_LaxWendroff, size * sizeof(float));
    hipMalloc((void**)&d_pn_FractionalStep, size * sizeof(float));
    hipMalloc((void**)&d_pn_MacCormack, size * sizeof(float));
    hipMalloc((void**)&d_pn_TVD, size * sizeof(float));
    hipMalloc((void**)&d_pn_PSOR, size * sizeof(float));
    hipMalloc((void**)&d_pn_FVS, size * sizeof(float));
    hipMalloc((void**)&d_v, size * sizeof(float));
    hipMalloc((void**)&d_pp_Galerkin, size * sizeof(float));
    hipMalloc((void**)&d_pp_Leapfrog, size * sizeof(float));
    hipMalloc((void**)&d_pp_CrankNicolson, size * sizeof(float));
    hipMalloc((void**)&d_pp_ADI, size * sizeof(float));
    hipMalloc((void**)&d_pp_Sigma, size * sizeof(float));
    hipMalloc((void**)&d_pp_LaxWendroff, size * sizeof(float));
    hipMalloc((void**)&d_pp_FractionalStep, size * sizeof(float));
    hipMalloc((void**)&d_pp_MacCormack, size * sizeof(float));
    hipMalloc((void**)&d_pp_TVD, size * sizeof(float));
    hipMalloc((void**)&d_pp_PSOR, size * sizeof(float));
    hipMalloc((void**)&d_pp_FVS, size * sizeof(float));

    // Transfer input data from host to device
    hipMemcpy(d_pn_Galerkin, h_pn_Galerkin, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_pn_Leapfrog, h_pn_Leapfrog, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_CrankNicolson, h_pn_CrankNicolson, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_ADI, h_pn_ADI, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_Sigma, h_pn_Sigma, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_LaxWendroff, h_pn_LaxWendroff, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_FractionalStep, h_pn_FractionalStep, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_MacCormack, h_pn_MacCormack, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_TVD, h_pn_TVD, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_PSOR, h_pn_PSOR, size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_pn_FVS, h_pn_FVS, size*sizeof(float),hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(8, 8);
    dim3 gridSize((nx + blockSize.x - 1) / blockSize.x);

    dim3 blockSize2(8);
    dim3 gridSize2((nx + blockSize2.x - 1) / blockSize2.x);

    // Current block size
    // int currentBlockSize = blockSize.x * blockSize.y * blockSize.z;
    int blockSizeLimit;
    hipDeviceGetAttribute(&blockSizeLimit, hipDeviceAttributeMaxThreadsPerBlock,0);
    printf("Max Threads Per Block: %d\n", blockSizeLimit);

    //=============================================================
    // Launch Galerkin Kernel and measure time
    //=============================================================
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1);

    Galerkin_2D_Solver<<<gridSize, blockSize>>>(nx, dx, nz, dz, dt, d_v, d_pn_Galerkin, d_pp_Galerkin);
    checkCUDAError("2D Galerkin Kernel launch");

    hipEventRecord(stop1);
    hipDeviceSynchronize();

    float time1 = 0;
    hipEventElapsedTime(&time1, start1, stop1);
    printf("Total Execution Time on GPU for 2D Galerkin kernel: %f ms\n", time1);

    // Transfer the result of Discontinuous_Galerkin_2D_Solver from device to host
    hipMemcpy(h_pp_Galerkin, d_pp_Galerkin, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the Galerkin_time array from device to host
    float GalerkinTime[N];
    hipMemcpyFromSymbol(GalerkinTime, HIP_SYMBOL(Galerkin_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of Galerkin elapsed time to a file1
    FILE *file1 = fopen("GalerkinTime_2D_data.txt", "w");
    if (file1 == NULL) {
    	fprintf(stderr, "Error opening GalerkinTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file1, "%.6f\n", GalerkinTime[i]);
    }
    fclose(file1);
/*
    // Save the result of Galerkin_2D_solver to a file_a
    FILE *file_a = fopen("Galerkin2DSolver.txt", "w");
    if (file_a == NULL) {
	    fprintf(stderr, "Error opening Galerkin2DSolver.txt file..\n");
	    return 1;
    }
    for (int j=0; j<size; j++) {
	    fprintf(file_a, "%.6f\n", h_pp_Galerkin[j]);
    }
    fclose(file_a);
*/

    //=============================================================
    // Launch Leapfrog Kernel and measure time
    //=============================================================

    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);

    Leapfrog_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_Leapfrog, d_pp_Leapfrog);
    checkCUDAError("2D Leapfrog Kernel launch");

    hipEventRecord(stop2);
    hipDeviceSynchronize();

    float time2 = 0;
    hipEventElapsedTime(&time2, start2, stop2);
    printf("Total Execution Time on GPU for 2D Leapfrog kernel: %f ms\n", time2);

    // Transfer the result of Leapfrog_2D_solver from device to host
    hipMemcpy(h_pp_Leapfrog, d_pp_Leapfrog, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the Leapfrog_time array from device to host
    float LeapfrogTime[N];
    hipMemcpyFromSymbol(LeapfrogTime, HIP_SYMBOL(Leapfrog_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of Leapfrog elapsed time to a file1
    FILE *file2 = fopen("LeapfrogTime_2D_data.txt", "w");
    if (file2 == NULL) {
    	fprintf(stderr, "Error opening LeapfrogTime_2D_data.txt file..\n");
	return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file2, "%.6f\n", LeapfrogTime[i]);
    }
    fclose(file2);
/*
    // Save the result of Leapfrog_2D_solver to a file_b
    FILE *file_b = fopen("Leapfrog2DSolver.txt", "w");
    if (file_b == NULL) {
	    fprintf(stderr, "Error opening Leapfrog2DSolver.txt file..\n");
	    return 1;
    }
    for (int k=0; k<size; k++) {
	    fprintf(file_b, "%.6f\n", h_pp_Leapfrog[k]);
    }
    fclose(file_b);
*/

    //=========================================================
    // Launch Crank-Nicolson kernel and measure time
    //=========================================================
    hipEvent_t start3, stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    hipEventRecord(start3);

    CrankNicolson_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_CrankNicolson, d_pp_CrankNicolson);
    checkCUDAError("2D CrankNicolson Kernel launch");

    hipEventRecord(stop3);
    hipDeviceSynchronize();

    float time3 = 0;
    hipEventElapsedTime(&time3, start3, stop3);
    printf("Total Execution Time on GPU for 2D CrankNicolson kernel: %f ms\n", time3);

    // Transfer the result of CrankNicolson_2D_solver from device to host
    hipMemcpy(h_pp_CrankNicolson, d_pp_CrankNicolson, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the CrankNicolson_time array from device to host
    float CrankNicolsonTime[N];
    hipMemcpyFromSymbol(CrankNicolsonTime, HIP_SYMBOL(CrankNicolson_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of CrankNicolson elapsed time to a file3
    FILE *file3 = fopen("CrankNicolsonTime_2D_data.txt", "w");
    if (file3 == NULL) {
    	fprintf(stderr, "Error opening CranknicolsonTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file3, "%.6f\n", CrankNicolsonTime[i]);
    }
    fclose(file3);
/*
    // Save the result of CrankNicolson_2D_solver to a file_c
    FILE *file_c = fopen("CrankNicolson2DSolver.txt", "w");
    if (file_c == NULL) {
	    fprintf(stderr, "Error opening CrankNicolson2DSolver.txt file..\n");
	    return 1;
    }
    for (int k=0; k<size; k++) {
	    fprintf(file_c, "%.6f\n", h_pp_CrankNicolson[k]);
    }
    fclose(file_c);
*/


    //=========================================================
    // Launch ADI kernel and measure time
    //=========================================================
    hipEvent_t start4, stop4;
    hipEventCreate(&start4);
    hipEventCreate(&stop4);
    hipEventRecord(start4);

    ADI_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_ADI, d_pp_ADI);
    checkCUDAError("2D ADI Kernel launch");

    hipEventRecord(stop4);
    hipDeviceSynchronize();

    float time4 = 0;
    hipEventElapsedTime(&time4, start4, stop4);
    printf("Total Execution Time on GPU for 2D ADI kernel: %f ms\n", time4);

    // Transfer the result of ADI_2D_solver from device to host
    hipMemcpy(h_pp_ADI, d_pp_ADI, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the ADI_time array from device to host
    float ADITime[N];
    hipMemcpyFromSymbol(ADITime, HIP_SYMBOL(ADI_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of ADI elapsed time to a file4
    FILE *file4 = fopen("ADITime_2D_data.txt", "w");
    if (file4 == NULL) {
    	fprintf(stderr, "Error opening ADITime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file4, "%.6f\n", ADITime[i]);
    }
    fclose(file4);
/*
    // Save the result of ADI_2D_solver to a file_d
    FILE *file_d = fopen("ADI2DSolver.txt", "w");
    if (file_d == NULL) {
	    fprintf(stderr, "Error opening ADI2DSolver.txt file..\n");
	    return 1;
    }
    for (int k=0; k<size; k++) {
	    fprintf(file_d, "%.6f\n", h_pp_ADI[k]);
    }
    fclose(file_d);
*/

    //=========================================================
    // Launch Sigma kernel and measure time
    //=========================================================
    hipEvent_t start5, stop5;
    hipEventCreate(&start5);
    hipEventCreate(&stop5);
    hipEventRecord(start5);

    Sigma_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_Sigma, d_pp_Sigma);
    checkCUDAError("2D Sigma Kernel launch");

    hipEventRecord(stop5);
    hipDeviceSynchronize();

    float time5 = 0;
    hipEventElapsedTime(&time5, start5, stop5);
    printf("Total Execution Time on GPU for 2D Sigma kernel: %f ms\n", time5);

    // Transfer the result of Sigma_2D_solver from device to host
    hipMemcpy(h_pp_Sigma, d_pp_Sigma, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the Sigma_time array from device to host
    float SigmaTime[N];
    hipMemcpyFromSymbol(SigmaTime, HIP_SYMBOL(Sigma_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of Sigma elapsed time to a file5
    FILE *file5 = fopen("SigmaTime_2D_data.txt", "w");
    if (file5 == NULL) {
    	fprintf(stderr, "Error opening SigmaTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file5, "%.6f\n", SigmaTime[i]);
    }
    fclose(file5);
/*
    // Save the result of Sigma_2D_solver to a file_e
    FILE *file_e = fopen("Sigma2DSolver.txt", "w");
    if (file_e == NULL) {
	    fprintf(stderr, "Error opening Sigma2DSolver.txt file..\n");
	    return 1;
    }
    for (int k=0; k<size; k++) {
	    fprintf(file_e, "%.6f\n", h_pp_Sigma[k]);
    }
    fclose(file_e);
*/

    //=========================================================
    // Launch LaxWendroff kernel and measure time
    //=========================================================
    hipEvent_t start6, stop6;
    hipEventCreate(&start6);
    hipEventCreate(&stop6);
    hipEventRecord(start6);

    LaxWendroff_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_LaxWendroff, d_pp_LaxWendroff);
    checkCUDAError("2D LaxWendroff Kernel launch");

    hipEventRecord(stop6);
    hipDeviceSynchronize();

    float time6 = 0;
    hipEventElapsedTime(&time6, start6, stop6);
    printf("Total Execution Time on GPU for 2D LaxWendroff kernel: %f ms\n", time6);

    // Transfer the result of LaxWendroff_2D_solver from device to host
    hipMemcpy(h_pp_LaxWendroff, d_pp_LaxWendroff, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the LaxWendroff_time array from device to host
    float LaxWendroffTime[N];
    hipMemcpyFromSymbol(LaxWendroffTime, HIP_SYMBOL(LaxWendroff_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of LaxWendroff elapsed time to a file6
    FILE *file6 = fopen("LaxWendroffTime_2D_data.txt", "w");
    if (file6 == NULL) {
    	fprintf(stderr, "Error opening LaxWendroffTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file6, "%.6f\n", LaxWendroffTime[i]);
    }
    fclose(file6);
/*
    // Save the result of LaxWendroff_2D_solver to a file_f
    FILE *file_f = fopen("LaxWendroff2DSolver.txt", "w");
    if (file_f == NULL) {
	    fprintf(stderr, "Error opening LaxWendroff2DSolver.txt file..\n");
	    return 1;
    }
    for (int k=0; k<size; k++) {
	    fprintf(file_f, "%.6f\n", h_pp_LaxWendroff[k]);
    }
    fclose(file_f);
*/

    //=========================================================
    // Launch Fractional Step kernel and measure time
    //=========================================================
    hipEvent_t start7, stop7;
    hipEventCreate(&start7);
    hipEventCreate(&stop7);
    hipEventRecord(start7);

    FractionalStep_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_FractionalStep, d_pp_FractionalStep);
    checkCUDAError("2D FractionalStep Kernel launch");

    hipEventRecord(stop7);
    hipDeviceSynchronize();

    float time7 = 0;
    hipEventElapsedTime(&time7, start7, stop7);
    printf("Total Execution Time on GPU for 2D FractionalStep kernel: %f ms\n", time7);

    // Transfer the result of FractionalStep_2D_solver from device to host
    hipMemcpy(h_pp_FractionalStep, d_pp_FractionalStep, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the FractionalStep_time array from device to host
    float FractionalStepTime[N];
    hipMemcpyFromSymbol(FractionalStepTime, HIP_SYMBOL(FractionalStep_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of FractionalStep elapsed time to a file7
    FILE *file7 = fopen("FractionalStepTime_2D_data.txt", "w");
    if (file7 == NULL) {
    	fprintf(stderr, "Error opening FractionalStepTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file7, "%.6f\n", FractionalStepTime[i]);
    }
    fclose(file7);
/*
    // Save the result of FractionalStep_2D_solver to a file_g
    FILE *file_g = fopen("FractionalStep2DSolver.txt", "w");
    if (file_g == NULL) {
	    fprintf(stderr, "Error opening FractionalStep2DSolver.txt file..\n");
	    return 1;
    }
    for (int k=0; k<size; k++) {
	    fprintf(file_g, "%.6f\n", h_pp_FractionalStep[k]);
    }
    fclose(file_g);
*/

    //=========================================================
    // Launch MacCormack kernel and measure time
    //=========================================================
    hipEvent_t start8, stop8;
    hipEventCreate(&start8);
    hipEventCreate(&stop8);
    hipEventRecord(start8);

    MacCormack_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_MacCormack, d_pp_MacCormack);
    checkCUDAError("2D MacCormack Kernel launch");

    hipEventRecord(stop8);
    hipDeviceSynchronize();

    float time8 = 0;
    hipEventElapsedTime(&time8, start8, stop8);
    printf("Total Execution Time on GPU for 2D MacCormack kernel: %f ms\n", time8);

    // Transfer the result of MacCormack_2D_solver from device to host
    hipMemcpy(h_pp_MacCormack, d_pp_MacCormack, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the MacCormack_time array from device to host
    float MacCormackTime[N];
    hipMemcpyFromSymbol(MacCormackTime, HIP_SYMBOL(MacCormack_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of MacCormack elapsed time to a file8
    FILE *file8 = fopen("MacCormackTime_2D_data.txt", "w");
    if (file8 == NULL) {
    	fprintf(stderr, "Error opening MacCormackTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file8, "%.6f\n", MacCormackTime[i]);
    }
    fclose(file8);
/*
    // Save the result of MacCormack_2D_solver to a file_h
    FILE *file_h = fopen("MacCormack2DSolver.txt", "w");
    if (file_h == NULL) {
	    fprintf(stderr, "Error opening MacCormack2DSolver.txt file..\n");
	    return 1;
    }
    for (int k=0; k<size; k++) {
	    fprintf(file_h, "%.6f\n", h_pp_MacCormack[k]);
    }
    fclose(file_h);
*/

    //=========================================================
    // Launch TVD kernel and measure time
    //=========================================================
    hipEvent_t start9, stop9;
    hipEventCreate(&start9);
    hipEventCreate(&stop9);
    hipEventRecord(start9);

    TVD_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_TVD, d_pp_TVD);
    checkCUDAError("2D TVD Kernel launch");

    hipEventRecord(stop9);
    hipDeviceSynchronize();

    float time9 = 0;
    hipEventElapsedTime(&time9, start9, stop9);
    printf("Total Execution Time on GPU for 2D TVD kernel: %f ms\n", time9);

    // Transfer the result of TVD_2D_solver from device to host
    hipMemcpy(h_pp_TVD, d_pp_TVD, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the TVD_time array from device to host
    float TVDTime[N];
    hipMemcpyFromSymbol(TVDTime, HIP_SYMBOL(TVD_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of TVD elapsed time to a file1
    FILE *file9 = fopen("TVDTime_2D_data.txt", "w");
    if (file9 == NULL) {
    	fprintf(stderr, "Error opening TVDTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file9, "%.6f\n", TVDTime[i]);
    }
    fclose(file9);
/*
    // Save the result of TVD_2D_solver to a file_c
    FILE *file_i = fopen("TVDSolver.txt", "w");
    if (file_i == NULL) {
	    fprintf(stderr, "Error opening TVD2DSolver.txt file..\n");
	    return 1;
    }
    for (int l=0; l<size; l++) {
	    fprintf(file_i, "%.6f\n", h_pp_TVD[l]);
    }
    fclose(file_i);
*/

    //=========================================================
    // Launch PSOR kernel and measure time
    //=========================================================
    hipEvent_t start10, stop10;
    hipEventCreate(&start10);
    hipEventCreate(&stop10);
    hipEventRecord(start10);

    PSOR_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_PSOR, d_pp_PSOR);
    checkCUDAError("2D PSOR Kernel launch");

    hipEventRecord(stop10);
    hipDeviceSynchronize();

    float time10 = 0;
    hipEventElapsedTime(&time10, start10, stop10);
    printf("Total Execution Time on GPU for 2D PSOR kernel: %f ms\n", time10);

    // Transfer the result of PSOR_2D_solver from device to host
    hipMemcpy(h_pp_PSOR, d_pp_PSOR, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the PSOR_time array from device to host
    float PSORTime[N];
    hipMemcpyFromSymbol(PSORTime, HIP_SYMBOL(PSOR_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of PSOR elapsed time to a file10
    FILE *file10 = fopen("PSORTime_2D_data.txt", "w");
    if (file10 == NULL) {
    	fprintf(stderr, "Error opening PSORTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file10, "%.6f\n", PSORTime[i]);
    }
    fclose(file10);
/*
    // Save the result of PSOR_2D_solver to a file_h
    FILE *file_j = fopen("PSOR2DSolver.txt", "w");
    if (file_j == NULL) {
	    fprintf(stderr, "Error opening PSOR2DSolver.txt file..\n");
	    return 1;
    }
    for (int l=0; l<size; l++) {
	    fprintf(file_j, "%.6f\n", h_pp_PSOR[l]);
    }
    fclose(file_j);
*/

    //=========================================================
    // Launch FVS kernel and measure time
    //=========================================================
    hipEvent_t start11, stop11;
    hipEventCreate(&start11);
    hipEventCreate(&stop11);
    hipEventRecord(start11);

    FVS_2D_Solver<<<gridSize2, blockSize2>>>(nx, dx, nz, dz, dt, d_v, d_pn_FVS, d_pp_FVS);
    checkCUDAError("2D FVS Kernel launch");

    hipEventRecord(stop11);
    hipDeviceSynchronize();

    float time11 = 0;
    hipEventElapsedTime(&time11, start11, stop11);
    printf("Total Execution Time on GPU for 2D FVS kernel: %f ms\n", time11);

    // Transfer the result of FVS_2D_solver from device to host
    hipMemcpy(h_pp_FVS, d_pp_FVS, size*sizeof(float),hipMemcpyDeviceToHost);

    // Transfer the FVS_time array from device to host
    float FVSTime[N];
    hipMemcpyFromSymbol(FVSTime, HIP_SYMBOL(FVS_time), N * sizeof(float), 0, hipMemcpyDeviceToHost);

    // Save the result of FVS elapsed time to a file10
    FILE *file11 = fopen("FVSTime_2D_data.txt", "w");
    if (file11 == NULL) {
    	fprintf(stderr, "Error opening FVSTime_2D_data.txt file..\n");
	    return 1;
    }
#pragma unroll
    for (int i=0; i < N; i++) {
    	fprintf(file11, "%.6f\n", FVSTime[i]);
    }
    fclose(file11);
/*
    // Save the result of FVS_2D_solver to a file_h
    FILE *file_k = fopen("FVS2DSolver.txt", "w");
    if (file_k == NULL) {
	    fprintf(stderr, "Error opening FVS2DSolver.txt file..\n");
	    return 1;
    }
    for (int l=0; l<size; l++) {
	    fprintf(file_k, "%.6f\n", h_pp_FVS[l]);
    }
    fclose(file_k);
*/

    hipEventDestroy(start1);
    hipEventDestroy(start2);
    hipEventDestroy(start3);
    hipEventDestroy(start4);
    hipEventDestroy(start5);
    hipEventDestroy(start6);
    hipEventDestroy(start7);
    hipEventDestroy(start8);
    hipEventDestroy(start9);
    hipEventDestroy(start10);
    hipEventDestroy(start11);
    hipEventDestroy(stop1);
    hipEventDestroy(stop2);
    hipEventDestroy(stop3);
    hipEventDestroy(stop4);
    hipEventDestroy(stop5);
    hipEventDestroy(stop6);
    hipEventDestroy(stop7);
    hipEventDestroy(stop8);
    hipEventDestroy(stop9);
    hipEventDestroy(stop10);
    hipEventDestroy(stop11);

    free(h_pn_Galerkin);
    free(h_pn_Leapfrog);
    free(h_pn_CrankNicolson);
    free(h_pn_ADI);
    free(h_pn_Sigma);
    free(h_pn_LaxWendroff);
    free(h_pn_FractionalStep);
    free(h_pn_MacCormack);
    free(h_pn_TVD);
    free(h_pn_PSOR);
    free(h_pn_FVS);
    free(h_v);
    free(h_pp_Galerkin);
    free(h_pp_Leapfrog);
    free(h_pp_CrankNicolson);
    free(h_pp_ADI);
    free(h_pp_Sigma);
    free(h_pp_LaxWendroff);
    free(h_pp_FractionalStep);
    free(h_pp_MacCormack);
    free(h_pp_TVD);
    free(h_pp_PSOR);
    free(h_pp_FVS);

    //free(GalerkinTime);
    //free(LeapfrogTime);
    //free(TVDTime);

    hipFree(d_pn_Galerkin);
    hipFree(d_pn_Leapfrog);
    hipFree(d_pn_CrankNicolson);
    hipFree(d_pn_ADI);
    hipFree(d_pn_Sigma);
    hipFree(d_pn_LaxWendroff);
    hipFree(d_pn_FractionalStep);
    hipFree(d_pn_MacCormack);
    hipFree(d_pn_TVD);
    hipFree(d_pn_PSOR);
    hipFree(d_pn_FVS);
    hipFree(d_v);
    hipFree(d_pp_Galerkin);
    hipFree(d_pp_Leapfrog);
    hipFree(d_pp_CrankNicolson);
    hipFree(d_pp_ADI);
    hipFree(d_pp_Sigma);
    hipFree(d_pp_LaxWendroff);
    hipFree(d_pp_FractionalStep);
    hipFree(d_pp_MacCormack);
    hipFree(d_pp_TVD);
    hipFree(d_pp_PSOR);
    hipFree(d_pp_FVS);

    return 0;
}






